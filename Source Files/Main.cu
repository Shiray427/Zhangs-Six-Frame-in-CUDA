#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <string>
#include <unordered_map>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""

#ifdef __INTELLISENSE__
#define __HIPCC__
#endif // __INTELLISENSE__
#include <hip/hip_cooperative_groups.h>
#ifdef __INTELLISENSE__
#undef __HIPCC__
#endif // __INTELLISENSE__

#include "../Header Files/gputimer.h"

using std::cin;
using std::cout;
using std::cerr;
using std::endl;
using std::string;
using std::unordered_map;
using std::ifstream;
using std::vector;
using std::milli;
using std::chrono::steady_clock;
using std::chrono::duration;
using std::to_string;
using std::copy;

namespace cg = cooperative_groups;

constexpr int gep = 2; // opening penalty
constexpr int gop = 3; // extend penalty
constexpr int shift = 4; // shift penalty
constexpr int infn = -999;
string myArray[40000][5];

int blosum62mat[24][24];
__device__ __constant__ int d_blosum62mat[24][24];
int score_top[4];

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)
template<typename T>
void check(T err, const char* const func, const char* const file, const int line)
{
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at: %s : %d\n", file, line);
        fprintf(stderr, "%s %s\n", hipGetErrorString(err), func);;
        exit(1);
    }
}

char DNA_to_Protein(string a) {
    unordered_map<string, char> DP{
        {"TTT", 'F'}, {"TTC", 'F'},
        {"TTA", 'L'}, {"TTG", 'L'},
        {"TCT", 'S'}, {"TCC", 'S'}, {"TCA", 'S'}, {"TCG", 'S'},
        {"TAT", 'Y'}, {"TAC", 'Y'},
        {"TGT", 'C'}, {"TGC", 'C'},
        {"TGG", 'W'},
        {"TAA", 'X'}, {"TAG", 'X'}, {"TGA", 'X'},
        {"CTT", 'L'}, {"CTC", 'L'}, {"CTA", 'L'}, {"CTG", 'L'},
        {"CCT", 'P'}, {"CCC", 'P'}, {"CCA", 'P'}, {"CCG", 'P'},
        {"CAT", 'H'}, {"CAC", 'H'},
        {"CAA", 'Q'}, {"CAG", 'Q'},
        {"CGA", 'R'}, {"CGT", 'R'}, {"CGC", 'R'}, {"CGG", 'R'},
        {"ATT", 'I'}, {"ATC", 'I'}, {"ATA",'I'},
        {"ATG", 'M'},
        {"ACT", 'T'}, {"ACA", 'T'}, {"ACG", 'T'}, {"ACC", 'T'},
        {"AAT", 'N'}, {"AAC", 'N'},
        {"AAG", 'K'}, {"AAA", 'K'},
        {"AGT", 'S'}, {"AGC", 'S'},
        {"AGA", 'R'}, {"AGG", 'R'},
        {"GTT", 'V'}, {"GTA", 'V'}, {"GTG", 'V'}, {"GTC", 'V'},
        {"GCT", 'A'}, {"GCC", 'A'}, {"GCG", 'A'}, {"GCA", 'A'},
        {"GAC", 'D'}, {"GAT", 'D'},
        {"GAA", 'E'}, {"GAG", 'E'},
        {"GGG", 'G'}, {"GGC", 'G'}, {"GGA", 'G'}, {"GGT", 'G'}
    };
    return DP[a];
}

__device__ char d_DNA_to_Protein(const char* dna_seq, int dna_index_1, int dna_index_2, int dna_index_3) {
    char codon[4] = "   ";
    codon[0] = dna_seq[dna_index_1];
    codon[1] = dna_seq[dna_index_2];
    codon[2] = dna_seq[dna_index_3];
    codon[3] = '\0';

    if (codon[0] == 'T') {
        if (codon[1] == 'T') {
            if (codon[2] == 'T' || codon[2] == 'C') return 'F';
            if (codon[2] == 'A' || codon[2] == 'G') return 'L';
        }
        else if (codon[1] == 'C') {
            return 'S';
        }
        else if (codon[1] == 'A') {
            if (codon[2] == 'T' || codon[2] == 'C') return 'Y';
            if (codon[2] == 'A' || codon[2] == 'G') return 'X';
        }
        else if (codon[1] == 'G') {
            if (codon[2] == 'T' || codon[2] == 'C') return 'C';
            if (codon[2] == 'G') return 'W';
            if (codon[2] == 'A') return 'X';
        }
    }
    else if (codon[0] == 'C') {
        if (codon[1] == 'T') return 'L';
        if (codon[1] == 'C') return 'P';
        if (codon[1] == 'A') {
            if (codon[2] == 'T' || codon[2] == 'C') return 'H';
            if (codon[2] == 'A' || codon[2] == 'G') return 'Q';
        }
        else if (codon[1] == 'G') return 'R';
    }
    else if (codon[0] == 'A') {
        if (codon[1] == 'T') {
            if (codon[2] == 'A' || codon[2] == 'C' || codon[2] == 'T') return 'I';
            if (codon[2] == 'G') return 'M';
        }
        if (codon[1] == 'C') return 'T';

        if (codon[1] == 'A') {
            if (codon[2] == 'T' || codon[2] == 'C') return 'N';
            if (codon[2] == 'G' || codon[2] == 'A') return 'K';

        }

        if (codon[1] == 'G') {
            if (codon[2] == 'T' || codon[2] == 'C') return 'S';
            if (codon[2] == 'A' || codon[2] == 'G') return 'R';
        }
    }
    else if (codon[0] == 'G') {
        if (codon[1] == 'T') return 'V';
        if (codon[1] == 'C') return 'A';
        if (codon[1] == 'A') {
            if (codon[2] == 'T' || codon[2] == 'C') return 'D';
            if (codon[2] == 'G' || codon[2] == 'A') return 'E';
        }
        if (codon[1] == 'G') return 'G';
    }
	return ' ';
}

string reverse_complement(string str) {
    unordered_map<char, char> RC{
        {'A', 'T'}, {'C', 'G'}, {'T', 'A'}, {'G', 'C'}
    };

    reverse(str.begin(), str.end());
    for (unsigned int i = 0; i < str.length(); i++) {
        str[i] = RC[str[i]];
    }
    return str;
}

void three_frame(string str, string* frame_one, string* frame_two, string* frame_three) {
    for (unsigned int i = 0; i < str.length(); i += 3) {
        if (i + 2 < str.length()) {
            *frame_one += DNA_to_Protein(str.substr(i, 3));
            *frame_two += DNA_to_Protein(str.substr(i + 1, 3));
            *frame_three += DNA_to_Protein(str.substr(i + 2, 3));
        }
    }
}

int place(char a) {
    unordered_map<char, int> blosumVal{
        {'A', 0}, {'R', 1}, {'N', 2}, {'D', 3}, {'C', 4}, {'Q', 5}, {'E', 6},
        {'G', 7}, {'H', 8}, {'I', 9}, {'L', 10}, {'K', 11 }, {'M', 12},
        {'F', 13}, {'P', 14}, {'S', 15}, {'T', 16}, {'W', 17}, {'Y', 18},
        {'V', 19}, {'B', 20}, {'Z', 21}, {'X', 22}, {'*', 23}
    };
    return blosumVal[(unsigned char)a];
}

__device__ int d_place(char a) {
    if (a == 'A') return 0;
    if (a == 'R') return 1;
    if (a == 'N') return 2;
    if (a == 'D') return 3;
    if (a == 'C') return 4;
    if (a == 'Q') return 5;
    if (a == 'E') return 6;
    if (a == 'G') return 7;
    if (a == 'H') return 8;
    if (a == 'I') return 9;
    if (a == 'L') return 10;
    if (a == 'K') return 11;
    if (a == 'M') return 12;
    if (a == 'F') return 13;
    if (a == 'P') return 14;
    if (a == 'S') return 15;
    if (a == 'T') return 16;
    if (a == 'W') return 17;
    if (a == 'Y') return 18;
    if (a == 'V') return 19;
    if (a == 'B') return 20;
    if (a == 'Z') return 21;
    if (a == 'X') return 22;
    if (a == '*') return 23;
    return -1;
}

int score(char a, char b) {
    int dA, dB;
    dA = place(a);
    dB = place(b);
    return blosum62mat[dA][dB];
}

__device__ int d_score(char a, char b) {
    int dA, dB;
    dA = d_place(a);
    dB = d_place(b);
    return d_blosum62mat[dA][dB];
}

void readBlosum62() {
    ifstream file("./Resource Files/BLOSUM62.txt");
	if (!file.is_open()) {
        cerr << "Error opening the BLOSUM62 file!" << endl;
		exit(1);
	}
    string skip;
    getline(file, skip);
    getline(file, skip);

    char c = NULL;

	for (int i = 0; i < 24; i++) {
		file >> c;
		for (int j = 0; j < 24; j++) {
			file >> blosum62mat[i][j];
		}
	}
	file.close();
}

vector<string> readFastaSequences(const string& filename) {
    ifstream file("./Resource Files/" + filename + ".fasta");
    vector<string> sequences;
    string line, sequence;

    if (!file) {
        cerr << "Error: Unable to open file " << filename << endl;
        return sequences;
    }

    while (getline(file, line)) {
        if (line.empty()) continue;
        if (line[0] == '>') {
            if (!sequence.empty()) {
                sequences.push_back(sequence);
                sequence.clear();
            }
        }
        else {
            sequence += line;
        }
    }

    if (!sequence.empty()) {
        sequences.push_back(sequence);
    }

    file.close();

    return sequences;
}

vector<string> readFastaIDs(const string& filename) {
    ifstream file("./Resource Files/" + filename + ".fasta");
    vector<string> sequenceIDs;
    string line;

    if (!file) {
        cerr << "Error: Unable to open file " << filename << endl;
        return sequenceIDs;
    }

    while (getline(file, line)) {
        if (!line.empty() && line[0] == '>') {
            sequenceIDs.push_back(line.substr(1)); // Remove '>' and store only the ID
        }
    }

    return sequenceIDs;
}

void init_local_v2(string input_seq, string ref_seq, int** sc_mat, int** ins_mat, int** del_mat, int** t_sc_mat, int** t_ins_mat, int** t_del_mat) {
    size_t N = input_seq.length();
    size_t M = ref_seq.length() + 1;

    for (size_t i = 0; i < N; i++) {
        ins_mat[i][0] = infn;
        t_ins_mat[i][0] = infn;
    }

    for (size_t i = 0; i < N; i++) {
        for (size_t j = 0; j < M; j++) {
            if (i == 0 || j == 0) {
                sc_mat[i][j] = 0;
            }
        }
    }

    for (size_t j = 0; j < M; j++) {
        del_mat[0][j] = infn;
        del_mat[2][j] = infn;
        del_mat[3][j] = infn;
        del_mat[1][j] = sc_mat[0][j] - gop - gep;

        t_del_mat[0][j] = infn;
        t_del_mat[2][j] = infn;
        t_del_mat[3][j] = infn;
        t_del_mat[1][j] = 1;
    }

    int insert = 0;
    int del = 0;
    int xscore = 0;
    int end = 3;

    for (size_t i = 0; i < 4; i++) {
        for (size_t j = 1; j < M; j++) {
            insert = ins_mat[i][j - 1] - gep;
            xscore = sc_mat[i][j - 1] - gop - gep;
            if (insert > xscore) {
                ins_mat[i][j] = insert;
            }
            else {
                ins_mat[i][j] = xscore;
            }

            insert = ins_mat[i][j];
            del = del_mat[i][j];

            if (i == 1) {
                xscore = sc_mat[0][j - 1] + score(DNA_to_Protein(input_seq.substr(i - 1, end)), ref_seq[j - 1]);
                if (insert >= del && insert >= xscore) {
                    sc_mat[i][j] = insert;
                }
                else if (del >= insert && del >= xscore) {
                    sc_mat[i][j] = del;
                }
                else {
                    sc_mat[i][j] = xscore;
                }

                if (sc_mat[i][j] == ins_mat[i][j]) {
                    t_sc_mat[i][j] = -2;
                }
                else if (sc_mat[i][j] == del_mat[i][j]) {
                    t_sc_mat[i][j] = -1;
                }
                else if (sc_mat[i][j] == xscore) {
                    t_sc_mat[i][j] = 1;
                }
            }
            else if (i == 2) {
                xscore = sc_mat[0][j - 1] + score(DNA_to_Protein(input_seq.substr(i - 1, end)), ref_seq[j - 1]) - shift;
                if (insert >= xscore) {
                    sc_mat[i][j] = insert;
                }
                else {
                    sc_mat[i][j] = xscore;
                }

                if (sc_mat[i][j] == ins_mat[i][j]) {
                    t_sc_mat[i][j] = -2;
                }
                else if (sc_mat[i][j] == xscore) {
                    t_sc_mat[i][j] = 2;
                }
            }
            else if (i == 3) {
                xscore = sc_mat[1][j - 1] + score(DNA_to_Protein(input_seq.substr(i - 1, end)), ref_seq[j - 1]) - shift;
                if (insert >= xscore) {
                    sc_mat[i][j] = insert;
                }
                else {
                    sc_mat[i][j] = xscore;
                }

                if (sc_mat[i][j] == ins_mat[i][j]) {
                    t_sc_mat[i][j] = -2;
                }
                else if (sc_mat[i][j] == xscore) {
                    t_sc_mat[i][j] = 2;
                }
            }

            if (sc_mat[i][j] < 0)
                sc_mat[i][j] = 0;
        }
    }
}

void init_local_v2_cuda(string input_seq, string ref_seq, int* u_sc_mat, int* u_ins_mat, int* u_del_mat, int* u_t_sc_mat, int* u_t_ins_mat, int* u_t_del_mat, size_t N, size_t M) {
    for (size_t i = 0; i < N; i++) {
        u_ins_mat[i * M] = infn;
        u_t_ins_mat[i * M] = infn;
    }

    for (size_t i = 0; i < N; i++) {
        for (size_t j = 0; j < M; j++) {
            if (i == 0 || j == 0) {
                u_sc_mat[i * M + j] = 0;
            }
        }
    }

    for (size_t j = 0; j < M; j++) {
        u_del_mat[0 * M + j] = infn;
        u_del_mat[2 * M + j] = infn;
        u_del_mat[3 * M + j] = infn;
        u_del_mat[1 * M + j] = u_sc_mat[0 * M + j] - gop - gep;

        u_t_del_mat[0 * M + j] = infn;
        u_t_del_mat[2 * M + j] = infn;
        u_t_del_mat[3 * M + j] = infn;
        u_t_del_mat[1 * M + j] = 1;
    }

    int insert = 0;
    int del = 0;
    int xscore = 0;
    int end = 3;

    for (int i = 0; i < 4; i++) {
        for (int j = 1; j < M; j++) {
            insert = u_ins_mat[i * M + (j - 1)] - gep;
            xscore = u_sc_mat[i * M + (j - 1)] - gop - gep;

            if (insert > xscore) {
                u_ins_mat[i * M + j] = insert;
            }
            else {
                u_ins_mat[i * M + j] = xscore;
            }

            insert = u_ins_mat[i * M + j];
            del = u_del_mat[i * M + j];

            if (i == 1) {
                xscore = u_sc_mat[0 * M + (j - 1)] + score(DNA_to_Protein(input_seq.substr(i - 1, end)), ref_seq[j - 1]);
                if (insert >= del && insert >= xscore) {
                    u_sc_mat[i * M + j] = insert;
                }
                else if (del >= insert && del >= xscore) {
                    u_sc_mat[i * M + j] = del;
                }
                else {
                    u_sc_mat[i * M + j] = xscore;
                }

                if (u_sc_mat[i * M + j] == u_ins_mat[i * M + j]) {
                    u_t_sc_mat[i * M + j] = -2;
                }
                else if (u_sc_mat[i * M + j] == u_del_mat[i * M + j]) {
                    u_t_sc_mat[i * M + j] = -1;
                }
                else if (u_sc_mat[i * M + j] == xscore) {
                    u_t_sc_mat[i * M + j] = 1;
                }
            }
            else if (i == 2) {
                xscore = u_sc_mat[0 * M + (j - 1)] + score(DNA_to_Protein(input_seq.substr(i - 1, end)), ref_seq[j - 1]) - shift;
                if (insert >= xscore) {
                    u_sc_mat[i * M + j] = insert;
                }
                else {
                    u_sc_mat[i * M + j] = xscore;
                }

                if (u_sc_mat[i * M + j] == u_ins_mat[i * M + j]) {
                    u_t_sc_mat[i * M + j] = -2;
                }
                else if (u_sc_mat[i * M + j] == xscore) {
                    u_t_sc_mat[i * M + j] = 2;
                }
            }
            else if (i == 3) {
                xscore = u_sc_mat[1 * M + (j - 1)] + score(DNA_to_Protein(input_seq.substr(i - 1, end)), ref_seq[j - 1]) - shift;
                if (insert >= xscore) {
                    u_sc_mat[i * M + j] = insert;
                }
                else {
                    u_sc_mat[i * M + j] = xscore;
                }

                if (u_sc_mat[i * M + j] == u_ins_mat[i * M + j]) {
                    u_t_sc_mat[i * M + j] = -2;
                }
                else if (u_sc_mat[i * M + j] == xscore) {
                    u_t_sc_mat[i * M + j] = 2;
                }
            }

            if (u_sc_mat[i * M + j] < 0) {
                u_sc_mat[i * M + j] = 0;
            }
        }
    }
}

void scoring_local_v2(string input_seq, string ref_seq, int** sc_mat, int** ins_mat, int** del_mat, int** t_sc_mat, int** t_ins_mat, int** t_del_mat) {
    size_t N = input_seq.length();
    size_t M = ref_seq.length() + 1;
    int insert = 0;
    int del = 0;
    int xscore = 0;
    int end = 3;
    int sc_1 = 0, sc_2 = 0, sc_3 = 0;
    int scoring = 0;
    char prot_seq;

    for (size_t i = 4; i < N; i++) {
        for (size_t j = 1; j < M; j++) {
            prot_seq = DNA_to_Protein(input_seq.substr(i - 1, end));
            scoring = score(prot_seq, ref_seq[j - 1]);
            insert = ins_mat[i][j - 1] - gep;
            xscore = sc_mat[i][j - 1] - gop - gep;
            if (insert > xscore) {
                ins_mat[i][j] = insert;
            }
            else {
                ins_mat[i][j] = xscore;
            }

            if (ins_mat[i][j] == insert) {
                t_ins_mat[i][j] = 0;
            }
            else if (ins_mat[i][j] == xscore) {
                t_ins_mat[i][j] = 1;
            }

            del = del_mat[i - 3][j] - gep;
            xscore = sc_mat[i - 3][j] - gop - gep;

            if (del >= xscore) {
                del_mat[i][j] = del;
            }
            else {
                del_mat[i][j] = xscore;
            }

            if (del_mat[i][j] == del) {
                t_del_mat[i][j] = 0;
            }
            else if (del_mat[i][j] == xscore) {
                t_del_mat[i][j] = 1;
            }

            if (i < N - 1) {
                insert = ins_mat[i][j];
                del = del_mat[i][j];
                sc_1 = sc_mat[i - 2][j - 1] + scoring - shift;
                sc_2 = sc_mat[i - 3][j - 1] + scoring;
                sc_3 = sc_mat[i - 4][j - 1] + scoring - shift;

                if (insert >= del && insert >= sc_1 && insert >= sc_2 && insert >= sc_3) {
                    sc_mat[i][j] = insert;
                }
                else if (del >= insert && del >= sc_1 && del >= sc_2 && del >= sc_3) {
                    sc_mat[i][j] = del;
                }
                else if (sc_1 >= insert && sc_1 >= del && sc_1 >= sc_2 && sc_1 >= sc_3) {
                    sc_mat[i][j] = sc_1;
                }
                else if (sc_2 >= insert && sc_2 >= del && sc_2 >= sc_1 && sc_2 >= sc_3) {
                    sc_mat[i][j] = sc_2;
                }
                else if (sc_3 >= insert && sc_3 >= del && sc_3 >= sc_1 && sc_3 >= sc_2) {
                    sc_mat[i][j] = sc_3;
                }

                if (sc_mat[i][j] == insert) {
                    t_sc_mat[i][j] = -2;
                }
                else if (sc_mat[i][j] == del) {
                    t_sc_mat[i][j] = -1;
                }
                else if (sc_mat[i][j] == sc_1) {
                    t_sc_mat[i][j] = 2;
                }
                else if (sc_mat[i][j] == sc_2) {
                    t_sc_mat[i][j] = 3;
                }
                else if (sc_mat[i][j] == sc_3) {
                    t_sc_mat[i][j] = 4;
                }
            }

            if (sc_mat[i][j] < 0) {
                sc_mat[i][j] = 0;
            }
        }
    }
    for (size_t i = N - 1; i < N; i++) {
        for (size_t j = 1; j < M; j++) {
            insert = ins_mat[i][j - 1] - gep;
            xscore = sc_mat[i][j - 1] - gop - gep;
            if (insert >= xscore) {
                ins_mat[i][j] = insert;
            }
            else {
                ins_mat[i][j] = xscore;
            }

            sc_mat[i][j] = infn;
            t_sc_mat[i][j] = infn;
        }
    }
}

__global__ void scoring_local_v2_cuda(const char* input_seq, const char* ref_seq, int* u_sc_mat, int* u_ins_mat, int* u_del_mat, int* u_t_sc_mat, int* u_t_ins_mat, int* u_t_del_mat, unsigned int N, unsigned int M, unsigned int submatrixStartX, unsigned int submatrixStartY, unsigned int submatrixSide) {
    cg::thread_block block = cg::this_thread_block();


    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    i += submatrixStartY;
    j += submatrixStartX;

    int insert = 0;
    int del = 0;
    int xscore = 0;
    int sc_1 = 0, sc_2 = 0, sc_3 = 0;
    int scoring = 0;
    char prot_seq;

    if (i >= 4 && i < N && j >= 1 && j < M) {
        for (unsigned int diag = 0; diag < 2 * submatrixSide - 1; ++diag) {
            if (i < submatrixStartY + submatrixSide && j < submatrixStartX + submatrixSide && (i - submatrixStartY) + (j - submatrixStartX) == diag) {
                prot_seq = d_DNA_to_Protein(input_seq, i - 1, i, i + 1);
                scoring = d_score(prot_seq, ref_seq[j - 1]);
                insert = u_ins_mat[i * M + (j - 1)] - gep;
                xscore = u_sc_mat[i * M + (j - 1)] - gop - gep;

                if (insert > xscore) {
                    u_ins_mat[i * M + j] = insert;
                }
                else {
                    u_ins_mat[i * M + j] = xscore;
                }

                if (u_ins_mat[i * M + j] == insert) {
                    u_t_ins_mat[i * M + j] = 0;
                }
                else if (u_ins_mat[i * M + j] == xscore) {
                    u_t_ins_mat[i * M + j] = 1;
                }

                del = u_del_mat[(i - 3) * M + j] - gep;
                xscore = u_sc_mat[(i - 3) * M + j] - gop - gep;

                if (del >= xscore) {
                    u_del_mat[i * M + j] = del;
                }
                else {
                    u_del_mat[i * M + j] = xscore;
                }

                if (u_del_mat[i * M + j] == del) {
                    u_t_del_mat[i * M + j] = 0;
                }
                else if (u_del_mat[i * M + j] == xscore) {
                    u_t_del_mat[i * M + j] = 1;
                }

                if (i < N - 1) {
                    insert = u_ins_mat[i * M + j];
                    del = u_del_mat[i * M + j];
                    sc_1 = u_sc_mat[(i - 2) * M + (j - 1)] + scoring - shift;
                    sc_2 = u_sc_mat[(i - 3) * M + (j - 1)] + scoring;
                    sc_3 = u_sc_mat[(i - 4) * M + (j - 1)] + scoring - shift;

                    if (insert >= del && insert >= sc_1 && insert >= sc_2 && insert >= sc_3) {
                        u_sc_mat[i * M + j] = insert;
                    }
                    else if (del >= insert && del >= sc_1 && del >= sc_2 && del >= sc_3) {
                        u_sc_mat[i * M + j] = del;
                    }
                    else if (sc_1 >= insert && sc_1 >= del && sc_1 >= sc_2 && sc_1 >= sc_3) {
                        u_sc_mat[i * M + j] = sc_1;
                    }
                    else if (sc_2 >= insert && sc_2 >= del && sc_2 >= sc_1 && sc_2 >= sc_3) {
                        u_sc_mat[i * M + j] = sc_2;
                    }
                    else if (sc_3 >= insert && sc_3 >= del && sc_3 >= sc_1 && sc_3 >= sc_2) {
                        u_sc_mat[i * M + j] = sc_3;
                    }

                    if (u_sc_mat[i * M + j] == insert) {
                        u_t_sc_mat[i * M + j] = -2;
                    }
                    else if (u_sc_mat[i * M + j] == del) {
                        u_t_sc_mat[i * M + j] = -1;
                    }
                    else if (u_sc_mat[i * M + j] == sc_1) {
                        u_t_sc_mat[i * M + j] = 2;
                    }
                    else if (u_sc_mat[i * M + j] == sc_2) {
                        u_t_sc_mat[i * M + j] = 3;
                    }
                    else if (u_sc_mat[i * M + j] == sc_3) {
                        u_t_sc_mat[i * M + j] = 4;
                    }
                }

                if (u_sc_mat[i * M + j] < 0) {
                    u_sc_mat[i * M + j] = 0;
                }
            }
            cg::sync(block);
        }
    }
    else if (i == N - 1 && j >= 1 && j < M) {
        for (unsigned int diag = 0; diag < 2 * submatrixSide - 1; ++diag) {
            if (i < submatrixStartY + submatrixSide && j < submatrixStartX + submatrixSide && (i - submatrixStartY) + (j - submatrixStartX) == diag) {
                insert = u_ins_mat[i * M + (j - 1)] - gep;
                xscore = u_sc_mat[i * M + (j - 1)] - gop - gep;
                if (insert >= xscore) {
                    u_ins_mat[i * M + j] = insert;
                }
                else {
                    u_ins_mat[i * M + j] = xscore;
                }

                u_sc_mat[i * M + j] = infn;
                u_t_sc_mat[i * M + j] = infn;
            }
        }
    }
}

void top5(int score, int index, int top_i, int top_j, int* score_top, int* top_i_max, int* top_j_max, int* top_indices) {
    // Check if the new score belongs in the top 5
    for (int x = 0; x < 5; x++) {
        if (score >= score_top[x]) {
            for (int y = 4; y > x; y--) {
                score_top[y] = score_top[y - 1];
                top_i_max[y] = top_i_max[y - 1];
                top_j_max[y] = top_j_max[y - 1];
                top_indices[y] = top_indices[y - 1];
            }

            score_top[x] = score;
            top_i_max[x] = top_i;
            top_j_max[x] = top_j;
            top_indices[x] = index;

            break;
        }
    }
}

void routine(int trace, int& i, int& j, string str, string ref_seq, string& final_seq1, string& final_seq2, string& frameshift) {
    int k = 0;
    if (trace == -2) {
        j--;
        k = i + 3;
        final_seq1 += "-";
        final_seq2 += ref_seq[j];
        frameshift += " ";
    }
    else if (trace == -1) {
        i -= 3;
        k = i + 3;
        final_seq1 += DNA_to_Protein(str.substr(k - 1, 3));;
        final_seq2 += "-";
        frameshift += " ";
    }
    else if (trace == 3) {
        i -= 3;
        j--;
        k = i + 3;
        final_seq1 += DNA_to_Protein(str.substr(k - 1, 3));
        final_seq2 += ref_seq[j];
        frameshift += " ";

    }
    else if (trace == 4) {
        i -= 4;
        j--;
        k = i + 4;
        final_seq1 += DNA_to_Protein(str.substr(k - 1, 3));
        final_seq2 += ref_seq[j];
        frameshift += "*";

    }
    else if (trace == 2) {
        i -= 2;
        j--;
        k = i + 2;
        final_seq1 += DNA_to_Protein(str.substr(k - 1, 3));
        final_seq2 += ref_seq[j];
        frameshift += "*";
    }
    else if (trace == 1) {
        i--;
        j--;
        k = i + 1;
        final_seq1 += DNA_to_Protein(str.substr(k - 1, 3));
        final_seq2 += ref_seq[j];
        frameshift += " ";
    }
}

//To be Replaced
void check_index(string input_dna, string ref_prot, string seq_dna, string seq_prot) {
    int i, high = -1, low = -1, hold = 0, temp = -1, curr = 0;
    for (i = 0; i < ref_prot.length(); i++) {
        if (ref_prot[i] == seq_prot[curr]) {
            if (curr == 0) {
                temp = i;
            }
            curr++;

            if (curr > hold) {
                hold = curr;
                low = temp;
                high = i;
            }
        }
        else {
            curr = 0;
            temp = -1;
        }

    }
    if (low != -1 && high != -1)
        cout << "Protein sequence indexes: " << low << " to " << high << endl;
}

void traceV2(string input_seq, string ref_seq, int** sc_mat, int** t_sc_mat, size_t N, size_t M, int index, int* indeces) {
    N = (int)N;
    M = (int)M;
    //indeces[3];
    int i_max = 0, j_max = 0, i = 0, j = 0, max_score = 0, curr_score;
    string f1, f2, f3, seq_dna, seq_prot, frameshift;
    three_frame(input_seq, &f1, &f2, &f3);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            curr_score = sc_mat[i][j];
            if (curr_score > max_score) {
                max_score = curr_score;
                i_max = i;
                j_max = j;
            }
        }
    }
    i = i_max;
    j = j_max;

    indeces[0] = max_score;
    indeces[1] = i_max;
    indeces[2] = j_max;


    while (sc_mat[i][j] != 0) {
        myArray[index][1] = to_string(j);
        routine(t_sc_mat[i][j], i, j, input_seq, ref_seq, seq_dna, seq_prot, frameshift);
    }


    cout << endl;
    reverse(seq_dna.begin(), seq_dna.end());
    reverse(seq_prot.begin(), seq_prot.end());
    reverse(frameshift.begin(), frameshift.end());

    cout << "frame 1: \t";
    for (i = 0; i < f1.length(); i++) {
        cout << f1[i] << "  ";
    }
    cout << endl;
    cout << "frame 2: \t ";
    for (i = 0; i < f2.length(); i++) {
        cout << f2[i] << "  ";
    }
    cout << endl;
    cout << "frame 3: \t  ";
    for (i = 0; i < f3.length(); i++) {
        cout << f3[i] << "  ";
    }
    cout << endl;

    cout << "Output DNA: \t";
    for (i = 0; i < seq_dna.length(); i++) {
        cout << seq_dna[i] << "  ";
    }
    cout << endl;
    cout << "Frameshift: \t";
    for (i = 0; i < frameshift.length(); i++) {
        cout << frameshift[i] << "  ";
    }
    cout << endl;
    cout << "Output Prot: \t";
    for (i = 0; i < seq_prot.length(); i++) {
        cout << seq_prot[i] << "  ";
    }
    cout << endl;
    cout << "Reference Seq: " << ref_seq << endl;
}

void traceV2_check(string input_seq, string ref_seq, int** sc_mat, int** t_sc_mat, size_t N, size_t M, int index, int* indeces) {
    N = (int)N;
    M = (int)M;
    int i_max = 0, j_max = 0, i = 0, j = 0, max_score = 0, curr_score;
    string f1, f2, f3, seq_dna, seq_prot, frameshift;
    three_frame(input_seq, &f1, &f2, &f3);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            curr_score = sc_mat[i][j];
            if (curr_score > max_score) {
                max_score = curr_score;
                i_max = i;
                j_max = j;
            }
        }
    }
    i = i_max;
    j = j_max;

    indeces[0] = max_score;
    indeces[1] = i_max;
    indeces[2] = j_max;

    myArray[index][0] = to_string(max_score);
    myArray[index][2] = to_string(j_max);

}

void traceV2_1d(string input_seq, string ref_seq, int* sc_mat, int* t_sc_mat, int N, int M, int index, int* indeces) {
    int i_max = 0, j_max = 0, i = 0, j = 0, max_score = 0, curr_score;
    string f1, f2, f3, seq_dna, seq_prot, frameshift;
    indeces[3];
    three_frame(input_seq, &f1, &f2, &f3);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            curr_score = sc_mat[i * M + j];
            if (curr_score > max_score) {
                max_score = curr_score;
                i_max = i;
                j_max = j;
            }
        }
    }

    i = i_max;
    j = j_max;

    indeces[0] = max_score;
    indeces[1] = i_max;
    indeces[2] = j_max;

    while (sc_mat[i * M + j] != 0) {

        int score_tc;
        score_tc = t_sc_mat[i * M + j];
        myArray[index][1] = to_string(j);
        routine(t_sc_mat[i * M + j], i, j, input_seq, ref_seq, seq_dna, seq_prot, frameshift);
    }

    cout << endl;
    reverse(seq_dna.begin(), seq_dna.end());
    reverse(seq_prot.begin(), seq_prot.end());
    reverse(frameshift.begin(), frameshift.end());

    cout << "frame 1: \t";
    for (i = 0; i < f1.length(); i++) {
        cout << f1[i] << "  ";
    }
    cout << endl;
    cout << "frame 2: \t ";
    for (i = 0; i < f2.length(); i++) {
        cout << f2[i] << "  ";
    }
    cout << endl;
    cout << "frame 3: \t  ";
    for (i = 0; i < f3.length(); i++) {
        cout << f3[i] << "  ";
    }
    cout << endl;

    cout << "Output DNA: \t";
    for (i = 0; i < seq_dna.length(); i++) {
        cout << seq_dna[i] << "  ";
    }
    cout << endl;
    cout << "Frameshift: \t";
    for (i = 0; i < frameshift.length(); i++) {
        cout << frameshift[i] << "  ";
    }
    cout << endl;
    cout << "Output Prot: \t";
    for (i = 0; i < seq_prot.length(); i++) {
        cout << seq_prot[i] << "  ";
    }
    cout << endl;
    cout << "Reference Seq: " << ref_seq << endl;
}

void traceV2_1d_check(string input_seq, string ref_seq, int* sc_mat, int* t_sc_mat, size_t N, size_t M, int index, int* indeces) {
    N = (int)N;
    M = (int)M;
    int i_max = 0, j_max = 0, i = 0, j = 0, max_score = 0, curr_score;
    string f1, f2, f3, seq_dna, seq_prot, frameshift;
    three_frame(input_seq, &f1, &f2, &f3);

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            curr_score = sc_mat[i * M + j];
            if (curr_score > max_score) {
                max_score = curr_score;
                i_max = i;
                j_max = j;
            }
        }
    }
    i = i_max;
    j = j_max;

    indeces[0] = max_score;
    indeces[1] = i_max;
    indeces[2] = j_max;

    myArray[index][0] = to_string(max_score);
    myArray[index][2] = to_string(j_max);

}


int main()
{
    int mode, frame, top, device;
    vector<string> dnaInputs, proteinInputs, proteinIdInputs;
    string protein_sequence, DNA_sequence, DNA_sequence_r, file_name;
    hipDeviceProp_t prop;
    GpuTimer timer;

    checkCudaErrors(hipGetDevice(&device));
    checkCudaErrors(hipSetDevice(device));

    checkCudaErrors(cudaGetDeviceProperties_v2(&prop, device));

    readBlosum62();

    hipMemcpyToSymbol(HIP_SYMBOL(d_blosum62mat), blosum62mat, sizeof(blosum62mat));

    do {
        file_name.clear();
        cout << "Input file name for protein(exclude .fasta/.fastq):" << endl << "--> ";
        cin >> file_name;

        proteinInputs = readFastaSequences(file_name);
        proteinIdInputs = readFastaIDs(file_name);
        if (!proteinInputs.empty() && !proteinIdInputs.empty()) {
            break;
        }
    } while (true);

    do {
        file_name.clear();
        cout << "Input file name for DNA (exclude .fasta/.fastq):" << endl << "--> ";
        cin >> file_name;

        dnaInputs = readFastaSequences(file_name);
        if (!dnaInputs.empty()) {
            break;
        }
    } while (true);

    do {
        cout << "Choose mode [0 - Sequential] [1 - CUDA]:" << endl << "--> ";
        cin >> mode;

        if (mode == 0 || mode == 1)
            break;

        cout << "Invalid input please try again." << endl;
    } while (true);

    do {
        cout << "Choose frame count [3 - 3 Frame] [6 - 6 Frame]:" << endl << "--> ";
        cin >> frame;

        if (frame == 3 || frame == 6) 
            break;
        
        cout << "Invalid input please try again." << endl;
    } while (true);

    do {
        cout << "Show Score [0 - Top 1] [1 - Top 5]:" << endl << "--> ";
        cin >> top;

        if (top == 0 || top == 1)
            break;

        cout << "Invalid input please try again." << endl;

    } while (true);

    for (int index_dna = 0; index_dna < dnaInputs.size(); index_dna++) {

        int* score_top1 = new int [5] {};
        int* top_scores = new int[5] {};
        int* top_i = new int[5] {};
        int* top_j = new int[5] {};
        int* top_indeces = new int[5] {};
        int* index = new int[4] {};
        int top_hold = 0;

        for (int index_prot = 0; index_prot < proteinInputs.size(); index_prot++) {

			DNA_sequence = dnaInputs[index_dna];
			protein_sequence = proteinInputs[index_prot];
            DNA_sequence_r = reverse_complement(DNA_sequence);

			size_t N = DNA_sequence.length();
			size_t M = protein_sequence.length() + 1;

            size_t N_size = N * sizeof(char);
            size_t M_size = M * sizeof(char);
			size_t size = N * M * sizeof(int);
            

            if (mode == 0) {

                int** sc_mat = new int* [N];
                int** ins_mat = new int* [N];
                int** del_mat = new int* [N];
                int** sc_mat_hold = new int* [N];

                int** t_sc_mat = new int* [N];
                int** t_ins_mat = new int* [N];
                int** t_del_mat = new int* [N];
                int** t_sc_mat_hold = new int* [N];

                for (int i = 0; i < N; i++) {
                    sc_mat[i] = new int[M]();
                    ins_mat[i] = new int[M]();
                    del_mat[i] = new int[M]();
                    sc_mat_hold[i] = new int[M]();

                    t_sc_mat[i] = new int[M]();
                    t_ins_mat[i] = new int[M]();
                    t_del_mat[i] = new int[M]();
                    t_sc_mat_hold[i] = new int[M]();
                }
                
				init_local_v2(DNA_sequence, protein_sequence, sc_mat, ins_mat, del_mat, t_sc_mat, t_ins_mat, t_del_mat);

                auto start = steady_clock::now();
				scoring_local_v2(DNA_sequence, protein_sequence, sc_mat, ins_mat, del_mat, t_sc_mat, t_ins_mat, t_del_mat);
                auto end = steady_clock::now();
                auto diff = end - start;

                traceV2_check(DNA_sequence, protein_sequence, sc_mat, t_sc_mat, N, M, index_prot, index);
                top5(index[0], index_prot, index[1], index[3], top_scores, top_i, top_j, top_indeces);
                cout << "Run DNA: " << index_dna << " Prot: " << index_prot << endl << "Time in ms: " << duration<double, milli>(diff).count() << endl;
                    
				if (frame == 6) {
					init_local_v2(DNA_sequence_r, protein_sequence, sc_mat, ins_mat, del_mat, t_sc_mat, t_ins_mat, t_del_mat);

                    auto start = steady_clock::now();
					scoring_local_v2(DNA_sequence_r, protein_sequence, sc_mat, ins_mat, del_mat, t_sc_mat, t_ins_mat, t_del_mat);
                    auto end = steady_clock::now();
                    auto diff = end - start;

                    traceV2_check(DNA_sequence_r, protein_sequence, sc_mat, t_sc_mat, N, M, index_prot, index);
                    top5(index[0], index_prot, index[1], index[3], top_scores, top_i, top_j, top_indeces);
                    cout << "Run DNA: " << index_dna << " Prot: " << index_prot << endl << "Time in ms: " << duration<double, milli>(diff).count() << endl;
				}

                for (int i = 0; i < N_size; i++) {
                    delete[] sc_mat[i];
                    delete[] ins_mat[i];
                    delete[] del_mat[i];
                    delete[] t_sc_mat[i];
                    delete[] t_ins_mat[i];
                    delete[] t_del_mat[i];
                    delete[] sc_mat_hold[i];
                    delete[] t_sc_mat_hold[i];
                }

                delete[] sc_mat;
                delete[] ins_mat;
                delete[] del_mat;
                delete[] t_sc_mat;
                delete[] t_ins_mat;
                delete[] t_del_mat;
                delete[] sc_mat_hold;
                delete[] t_sc_mat_hold;
            }
            else if (mode == 1) {

                char* d_DNA_sequence;
                char* d_protein_sequence;
                char* d_DNA_sequence_r;

                int* u_sc_mat;
                int* u_ins_mat;
                int* u_del_mat;

                int* u_t_sc_mat;
                int* u_t_ins_mat;
                int* u_t_del_mat;

                checkCudaErrors(hipMalloc(&d_DNA_sequence, N_size));
                checkCudaErrors(hipMalloc(&d_protein_sequence, M_size));
                checkCudaErrors(hipMalloc(&d_DNA_sequence_r, N_size));

                checkCudaErrors(hipMemcpy(d_DNA_sequence, DNA_sequence.c_str(), N_size, hipMemcpyHostToDevice));
                checkCudaErrors(hipMemcpy(d_protein_sequence, protein_sequence.c_str(), M_size, hipMemcpyHostToDevice));
                checkCudaErrors(hipMemcpy(d_DNA_sequence_r, DNA_sequence_r.c_str(), N_size, hipMemcpyHostToDevice));

                checkCudaErrors(hipMallocManaged(&u_sc_mat, size));
                checkCudaErrors(hipMallocManaged(&u_ins_mat, size));
                checkCudaErrors(hipMallocManaged(&u_del_mat, size));

                checkCudaErrors(hipMallocManaged(&u_t_sc_mat, size));
                checkCudaErrors(hipMallocManaged(&u_t_ins_mat, size));
                checkCudaErrors(hipMallocManaged(&u_t_del_mat, size));

                checkCudaErrors(hipMemset(u_sc_mat, 0, size));
                checkCudaErrors(hipMemset(u_ins_mat, 0, size));
                checkCudaErrors(hipMemset(u_del_mat, 0, size));

                checkCudaErrors(hipMemset(u_t_sc_mat, 0, size));
                checkCudaErrors(hipMemset(u_t_ins_mat, 0, size));
                checkCudaErrors(hipMemset(u_t_del_mat, 0, size));

                dim3 blockDimMain(32, 32);
                dim3 gridDimMain(1);
				dim3 blockDimLastRow(1024);
                dim3 gridDimLastRow(((unsigned int)(M - 1) + blockDimLastRow.x - 1) / blockDimLastRow.x);


				unsigned int submatrixSide = blockDimMain.x;
				unsigned int numSubmatrixRows = ((unsigned int)N + submatrixSide - 1) / submatrixSide;
				unsigned int numSubmatrixCols = ((unsigned int)M + submatrixSide - 1) / submatrixSide;

				init_local_v2_cuda(DNA_sequence, protein_sequence, u_sc_mat, u_ins_mat, u_del_mat, u_t_sc_mat, u_t_ins_mat, u_t_del_mat, N, M);

                timer.Start();
                for (unsigned int diag = 0; diag < numSubmatrixRows + numSubmatrixCols - 1; ++diag) {
                    for (unsigned int submatrixY = std::max(0, (int)diag - (int)(numSubmatrixCols - 1)); submatrixY <= diag && submatrixY < numSubmatrixRows; ++submatrixY) {
                        int submatrixX = diag - submatrixY;
                        scoring_local_v2_cuda << <gridDimMain, blockDimMain >> > (d_DNA_sequence, d_protein_sequence, u_sc_mat, u_ins_mat, u_del_mat, u_t_sc_mat, u_t_ins_mat, u_t_del_mat, N, M, submatrixX * submatrixSide, submatrixY * submatrixSide, submatrixSide);
                        checkCudaErrors(hipGetLastError());
                    }
                    checkCudaErrors(hipDeviceSynchronize());
                }
                timer.Stop();        

                traceV2_1d_check(DNA_sequence, protein_sequence, u_sc_mat, u_t_sc_mat, N, M, index_prot, index);
                cout << "Run DNA: " << index_dna << " Prot: " << index_prot << endl << "Time in ms: " << timer.Elapsed() << endl;
                top5(index[0], index_prot, index[1], index[2], top_scores, top_i, top_j, top_indeces);
   
				if (frame == 6) {
					init_local_v2_cuda(DNA_sequence_r, protein_sequence, u_sc_mat, u_ins_mat, u_del_mat, u_t_sc_mat, u_t_ins_mat, u_t_del_mat, N, M);
						
                    timer.Start();
                    for (unsigned int diag = 0; diag < numSubmatrixRows + numSubmatrixCols - 1; ++diag) {
                        for (unsigned int submatrixY = std::max(0, (int)diag - (int)(numSubmatrixCols - 1)); submatrixY <= diag && submatrixY < numSubmatrixRows; ++submatrixY) {
                            int submatrixX = diag - submatrixY;
                            scoring_local_v2_cuda << <gridDimMain, blockDimMain >> > (d_DNA_sequence_r, d_protein_sequence, u_sc_mat, u_ins_mat, u_del_mat, u_t_sc_mat, u_t_ins_mat, u_t_del_mat, N, M, submatrixX * submatrixSide, submatrixY * submatrixSide, submatrixSide);
                            checkCudaErrors(hipGetLastError());
                        }
                        checkCudaErrors(hipDeviceSynchronize());
                    }
					timer.Stop();

                    traceV2_1d_check(DNA_sequence_r, protein_sequence, u_sc_mat, u_t_sc_mat, N, M, index_prot, index);
                    cout << "Run DNA: " << index_dna << " Prot: " << index_prot << endl << "Time in ms: " << timer.Elapsed() << endl;
                    top5(index[0], index_prot, index[1], index[2], top_scores, top_i, top_j, top_indeces);
				}
				

                checkCudaErrors(hipFree(d_DNA_sequence));
                checkCudaErrors(hipFree(d_protein_sequence));
                checkCudaErrors(hipFree(d_DNA_sequence_r));

                checkCudaErrors(hipFree(u_sc_mat));
                checkCudaErrors(hipFree(u_ins_mat));
                checkCudaErrors(hipFree(u_del_mat));

                checkCudaErrors(hipFree(u_t_sc_mat));
                checkCudaErrors(hipFree(u_t_ins_mat));
                checkCudaErrors(hipFree(u_t_del_mat));
            }
        }

        if (top == 0) {
            for (int i = 0; i < 5; i++) {
                DNA_sequence = dnaInputs[index_dna];
                protein_sequence = proteinInputs[top_indeces[i]];
                DNA_sequence_r = reverse_complement(DNA_sequence);

                size_t N = DNA_sequence.length();
                size_t M = protein_sequence.length() + 1;

                size_t N_size = N * sizeof(char);
                size_t M_size = M * sizeof(char);
                size_t size = N * M * sizeof(int);
                top_hold = top_scores[0];

                if (mode == 0 && top_hold == top_scores[i]) {
                    cout << proteinIdInputs[top_indeces[i]] << endl << "Sequence: " << top_indeces[i] << endl;
                    int** sc_mat = new int* [N];
                    int** ins_mat = new int* [N];
                    int** del_mat = new int* [N];
                    int** sc_mat_hold = new int* [N];

                    int** t_sc_mat = new int* [N];
                    int** t_ins_mat = new int* [N];
                    int** t_del_mat = new int* [N];
                    int** t_sc_mat_hold = new int* [N];

                    for (int i = 0; i < N; i++) {
                        sc_mat[i] = new int[M]();
                        ins_mat[i] = new int[M]();
                        del_mat[i] = new int[M]();
                        sc_mat_hold[i] = new int[M]();

                        t_sc_mat[i] = new int[M]();
                        t_ins_mat[i] = new int[M]();
                        t_del_mat[i] = new int[M]();
                        t_sc_mat_hold[i] = new int[M]();
                    }

                    init_local_v2(DNA_sequence, protein_sequence, sc_mat, ins_mat, del_mat, t_sc_mat, t_ins_mat, t_del_mat);
                    auto start = steady_clock::now();
                    scoring_local_v2(DNA_sequence, protein_sequence, sc_mat, ins_mat, del_mat, t_sc_mat, t_ins_mat, t_del_mat);
                    auto end = steady_clock::now();
                    auto diff = end - start;
                    traceV2(DNA_sequence, protein_sequence, sc_mat, t_sc_mat, N, M, top_indeces[i], index);
                    cout << endl << "Score: " << top_scores[i] << endl;
                    cout << "Start to End match in Protein: " << myArray[top_indeces[i]][1] << "-" << myArray[top_indeces[i]][2] << endl << endl;
                    cout << "Time in ms: " << duration<double, milli>(diff).count() << endl << endl;

                    for (int i = 0; i < N_size; i++) {
                        delete[] sc_mat[i];
                        delete[] ins_mat[i];
                        delete[] del_mat[i];
                        delete[] t_sc_mat[i];
                        delete[] t_ins_mat[i];
                        delete[] t_del_mat[i];
                        delete[] sc_mat_hold[i];
                        delete[] t_sc_mat_hold[i];
                    }

                    delete[] sc_mat;
                    delete[] ins_mat;
                    delete[] del_mat;
                    delete[] t_sc_mat;
                    delete[] t_ins_mat;
                    delete[] t_del_mat;
                    delete[] sc_mat_hold;
                    delete[] t_sc_mat_hold;

                }
                else if (mode == 1 && top_hold == top_scores[i]) {
                    cout << proteinIdInputs[top_indeces[i]] << endl << "Sequence: " << top_indeces[i] << endl;
                    char* d_DNA_sequence;
                    char* d_protein_sequence;
                    char* d_DNA_sequence_r;

                    int* u_sc_mat;
                    int* u_ins_mat;
                    int* u_del_mat;

                    int* u_t_sc_mat;
                    int* u_t_ins_mat;
                    int* u_t_del_mat;

                    checkCudaErrors(hipMalloc(&d_DNA_sequence, N_size));
                    checkCudaErrors(hipMalloc(&d_protein_sequence, M_size));
                    checkCudaErrors(hipMalloc(&d_DNA_sequence_r, N_size));

                    checkCudaErrors(hipMemcpy(d_DNA_sequence, DNA_sequence.c_str(), N_size, hipMemcpyHostToDevice));
                    checkCudaErrors(hipMemcpy(d_protein_sequence, protein_sequence.c_str(), M_size, hipMemcpyHostToDevice));
                    checkCudaErrors(hipMemcpy(d_DNA_sequence_r, DNA_sequence_r.c_str(), N_size, hipMemcpyHostToDevice));

                    checkCudaErrors(hipMallocManaged(&u_sc_mat, size));
                    checkCudaErrors(hipMallocManaged(&u_ins_mat, size));
                    checkCudaErrors(hipMallocManaged(&u_del_mat, size));

                    checkCudaErrors(hipMallocManaged(&u_t_sc_mat, size));
                    checkCudaErrors(hipMallocManaged(&u_t_ins_mat, size));
                    checkCudaErrors(hipMallocManaged(&u_t_del_mat, size));

                    checkCudaErrors(hipMemset(u_sc_mat, 0, size));
                    checkCudaErrors(hipMemset(u_ins_mat, 0, size));
                    checkCudaErrors(hipMemset(u_del_mat, 0, size));

                    checkCudaErrors(hipMemset(u_t_sc_mat, 0, size));
                    checkCudaErrors(hipMemset(u_t_ins_mat, 0, size));
                    checkCudaErrors(hipMemset(u_t_del_mat, 0, size));

                    dim3 blockDimMain(32, 32);
                    dim3 gridDimMain(1);
                    dim3 blockDimLastRow(1024);
                    dim3 gridDimLastRow(((unsigned int)(M - 1) + blockDimLastRow.x - 1) / blockDimLastRow.x);


                    unsigned int submatrixSide = blockDimMain.x;
                    unsigned int numSubmatrixRows = ((unsigned int)N + submatrixSide - 1) / submatrixSide;
                    unsigned int numSubmatrixCols = ((unsigned int)M + submatrixSide - 1) / submatrixSide;

                    init_local_v2_cuda(DNA_sequence, protein_sequence, u_sc_mat, u_ins_mat, u_del_mat, u_t_sc_mat, u_t_ins_mat, u_t_del_mat, N, M);

                    timer.Start();
                    for (unsigned int diag = 0; diag < numSubmatrixRows + numSubmatrixCols - 1; ++diag) {
                        for (unsigned int submatrixY = std::max(0, (int)diag - (int)(numSubmatrixCols - 1)); submatrixY <= diag && submatrixY < numSubmatrixRows; ++submatrixY) {
                            int submatrixX = diag - submatrixY;
                            scoring_local_v2_cuda << <gridDimMain, blockDimMain >> > (d_DNA_sequence, d_protein_sequence, u_sc_mat, u_ins_mat, u_del_mat, u_t_sc_mat, u_t_ins_mat, u_t_del_mat, N, M, submatrixX * submatrixSide, submatrixY * submatrixSide, submatrixSide);
                            checkCudaErrors(hipGetLastError());
                        }
                        checkCudaErrors(hipDeviceSynchronize());
                    }
                    timer.Stop();
                    traceV2_1d(DNA_sequence, protein_sequence, u_sc_mat, u_t_sc_mat, N, M, top_indeces[i], index);
                    cout << endl << "Score: " << top_scores[i] << endl;
                    cout << "Start to End match in Protein: " << myArray[top_indeces[i]][1] << "-" << myArray[top_indeces[i]][2] << endl << endl;
                    cout << "Time in ms: " << timer.Elapsed() << endl << endl;

                    checkCudaErrors(hipFree(d_DNA_sequence));
                    checkCudaErrors(hipFree(d_protein_sequence));
                    checkCudaErrors(hipFree(d_DNA_sequence_r));

                    checkCudaErrors(hipFree(u_sc_mat));
                    checkCudaErrors(hipFree(u_ins_mat));
                    checkCudaErrors(hipFree(u_del_mat));

                    checkCudaErrors(hipFree(u_t_sc_mat));
                    checkCudaErrors(hipFree(u_t_ins_mat));
                    checkCudaErrors(hipFree(u_t_del_mat));

                }
            }
        }
        else if (top == 1) {
            for (int i = 0; i < 5; i++) {
                cout << proteinIdInputs[top_indeces[i]] << endl << "Sequence: " << top_indeces[i] << endl;
                DNA_sequence = dnaInputs[index_dna];
                protein_sequence = proteinInputs[top_indeces[i]];
                DNA_sequence_r = reverse_complement(DNA_sequence);

                size_t N = DNA_sequence.length();
                size_t M = protein_sequence.length() + 1;

                size_t N_size = N * sizeof(char);
                size_t M_size = M * sizeof(char);
                size_t size = N * M * sizeof(int);

                if (mode == 0) {

                    int** sc_mat = new int* [N];
                    int** ins_mat = new int* [N];
                    int** del_mat = new int* [N];
                    int** sc_mat_hold = new int* [N];

                    int** t_sc_mat = new int* [N];
                    int** t_ins_mat = new int* [N];
                    int** t_del_mat = new int* [N];
                    int** t_sc_mat_hold = new int* [N];

                    for (int i = 0; i < N; i++) {
                        sc_mat[i] = new int[M]();
                        ins_mat[i] = new int[M]();
                        del_mat[i] = new int[M]();
                        sc_mat_hold[i] = new int[M]();

                        t_sc_mat[i] = new int[M]();
                        t_ins_mat[i] = new int[M]();
                        t_del_mat[i] = new int[M]();
                        t_sc_mat_hold[i] = new int[M]();
                    }

                    init_local_v2(DNA_sequence, protein_sequence, sc_mat, ins_mat, del_mat, t_sc_mat, t_ins_mat, t_del_mat);
                    auto start = steady_clock::now();
                    scoring_local_v2(DNA_sequence, protein_sequence, sc_mat, ins_mat, del_mat, t_sc_mat, t_ins_mat, t_del_mat);
                    auto end = steady_clock::now();
                    auto diff = end - start;
                    traceV2(DNA_sequence, protein_sequence, sc_mat, t_sc_mat, N, M, top_indeces[i], index);
                    cout << endl << "Score: " << top_scores[i] << endl;
                    cout << "Start to End match in Protein: " << myArray[top_indeces[i]][1] << "-" << myArray[top_indeces[i]][2] << endl << endl;
                    cout << "Time in ms: " << duration<double, milli>(diff).count() << endl << endl;

                    for (int i = 0; i < N_size; i++) {
                        delete[] sc_mat[i];
                        delete[] ins_mat[i];
                        delete[] del_mat[i];
                        delete[] t_sc_mat[i];
                        delete[] t_ins_mat[i];
                        delete[] t_del_mat[i];
                        delete[] sc_mat_hold[i];
                        delete[] t_sc_mat_hold[i];
                    }

                    delete[] sc_mat;
                    delete[] ins_mat;
                    delete[] del_mat;
                    delete[] t_sc_mat;
                    delete[] t_ins_mat;
                    delete[] t_del_mat;
                    delete[] sc_mat_hold;
                    delete[] t_sc_mat_hold;

                }
                else if (mode == 1) {

                    char* d_DNA_sequence;
                    char* d_protein_sequence;
                    char* d_DNA_sequence_r;

                    int* u_sc_mat;
                    int* u_ins_mat;
                    int* u_del_mat;

                    int* u_t_sc_mat;
                    int* u_t_ins_mat;
                    int* u_t_del_mat;

                    checkCudaErrors(hipMalloc(&d_DNA_sequence, N_size));
                    checkCudaErrors(hipMalloc(&d_protein_sequence, M_size));
                    checkCudaErrors(hipMalloc(&d_DNA_sequence_r, N_size));

                    checkCudaErrors(hipMemcpy(d_DNA_sequence, DNA_sequence.c_str(), N_size, hipMemcpyHostToDevice));
                    checkCudaErrors(hipMemcpy(d_protein_sequence, protein_sequence.c_str(), M_size, hipMemcpyHostToDevice));
                    checkCudaErrors(hipMemcpy(d_DNA_sequence_r, DNA_sequence_r.c_str(), N_size, hipMemcpyHostToDevice));

                    checkCudaErrors(hipMallocManaged(&u_sc_mat, size));
                    checkCudaErrors(hipMallocManaged(&u_ins_mat, size));
                    checkCudaErrors(hipMallocManaged(&u_del_mat, size));

                    checkCudaErrors(hipMallocManaged(&u_t_sc_mat, size));
                    checkCudaErrors(hipMallocManaged(&u_t_ins_mat, size));
                    checkCudaErrors(hipMallocManaged(&u_t_del_mat, size));

                    checkCudaErrors(hipMemset(u_sc_mat, 0, size));
                    checkCudaErrors(hipMemset(u_ins_mat, 0, size));
                    checkCudaErrors(hipMemset(u_del_mat, 0, size));

                    checkCudaErrors(hipMemset(u_t_sc_mat, 0, size));
                    checkCudaErrors(hipMemset(u_t_ins_mat, 0, size));
                    checkCudaErrors(hipMemset(u_t_del_mat, 0, size));

                    dim3 blockDimMain(32, 32);
                    dim3 gridDimMain(1);
                    dim3 blockDimLastRow(1024);
                    dim3 gridDimLastRow(((unsigned int)(M - 1) + blockDimLastRow.x - 1) / blockDimLastRow.x);


                    unsigned int submatrixSide = blockDimMain.x;
                    unsigned int numSubmatrixRows = ((unsigned int)N + submatrixSide - 1) / submatrixSide;
                    unsigned int numSubmatrixCols = ((unsigned int)M + submatrixSide - 1) / submatrixSide;

                    init_local_v2_cuda(DNA_sequence, protein_sequence, u_sc_mat, u_ins_mat, u_del_mat, u_t_sc_mat, u_t_ins_mat, u_t_del_mat, N, M);

                    timer.Start();
                    for (unsigned int diag = 0; diag < numSubmatrixRows + numSubmatrixCols - 1; ++diag) {
                        for (unsigned int submatrixY = std::max(0, (int)diag - (int)(numSubmatrixCols - 1)); submatrixY <= diag && submatrixY < numSubmatrixRows; ++submatrixY) {
                            int submatrixX = diag - submatrixY;
                            scoring_local_v2_cuda << <gridDimMain, blockDimMain >> > (d_DNA_sequence, d_protein_sequence, u_sc_mat, u_ins_mat, u_del_mat, u_t_sc_mat, u_t_ins_mat, u_t_del_mat, N, M, submatrixX * submatrixSide, submatrixY * submatrixSide, submatrixSide);
                            checkCudaErrors(hipGetLastError());
                        }
                        checkCudaErrors(hipDeviceSynchronize());
                    }
                    timer.Stop();

                    traceV2_1d(DNA_sequence, protein_sequence, u_sc_mat, u_t_sc_mat, N, M, top_indeces[i], index);
                    cout << endl << "Score: " << top_scores[i] << endl;
                    cout << "Start to End match in Protein: " << myArray[top_indeces[i]][1] << "-" << myArray[top_indeces[i]][2] << endl << endl;
                    cout << "Time in ms: " << timer.Elapsed() << endl << endl;

                    checkCudaErrors(hipFree(d_DNA_sequence));
                    checkCudaErrors(hipFree(d_protein_sequence));
                    checkCudaErrors(hipFree(d_DNA_sequence_r));

                    checkCudaErrors(hipFree(u_sc_mat));
                    checkCudaErrors(hipFree(u_ins_mat));
                    checkCudaErrors(hipFree(u_del_mat));

                    checkCudaErrors(hipFree(u_t_sc_mat));
                    checkCudaErrors(hipFree(u_t_ins_mat));
                    checkCudaErrors(hipFree(u_t_del_mat));

                }
            }

        }
    }

    checkCudaErrors(hipDeviceReset());

    return 0;
}